#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "../common/timer.h"
#include "../common/cuda_help.h"

#define MAX_VAL         (255)
#define KERNEL_SIZE     (3)
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define WRAP_INDEX(idx, max) (((idx) + (max)) % (max))
// #define DEBUG

int num_blocks = 2;
int threads_per_block = 4 * MIN_CUDA_THREADS;

void print_arr(double* data, size_t N);

__global__ void init_arr(double *data_d, int n, unsigned long seed) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState state;
    hiprand_init(seed, index, 0, &state);

    for (int i = index; i < n; i += stride) {
        data_d[i] = (double)(hiprand_uniform(&state) * INT_MAX);
        // data_d[i] = i + 1;
    }
}

__global__ void neighbor_kernel(double* data_d, size_t n) {
    size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    size_t stride = 2 * (blockDim.x * gridDim.x);

    for (size_t i = index; i < n; i += stride) {
        if (i + 1 < n) {
            data_d[i / 2] = data_d[i] + data_d[i + 1];
        } else {
            data_d[i / 2] = data_d[i];  
        }
    }
}

double reduction_neighbor(double* data_d, size_t N) {
    while (N > 1) {
        size_t newN = (N + 1) / 2;
        neighbor_kernel<<<num_blocks, threads_per_block>>>(data_d, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
        N = newN;
    }

    double val;
    CHECK_CUDA(hipMemcpy(&val, data_d, sizeof(double), hipMemcpyDeviceToHost));

    return val;
}

__global__ void stride_kernel(double* data_d, size_t n) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t window_stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n; i += window_stride) {
        size_t stride = (n + 1) / 2;
        if (i + stride < n) {
            data_d[i] += data_d[i + stride];
        }
    }
}

double reduction_stride(double* data_d, size_t N) {
    while (N > 1) {
        size_t newN = (N + 1) / 2;
        stride_kernel<<<num_blocks, threads_per_block>>>(data_d, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
        N = newN;
    }

    double val;
    CHECK_CUDA(hipMemcpy(&val, data_d, sizeof(double), hipMemcpyDeviceToHost));

    return val;
}

__global__ void seg_scan_red_kernel(double* data_d, size_t n, size_t stride) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t window_stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n; i += window_stride) {
        size_t target_idx = (i + 1) * (2 * stride) - 1;

        if (target_idx < n) { 
            data_d[target_idx] += data_d[target_idx - stride];
        }
    }
}

__global__ void seg_scan_ds_kernel(double* data_d, size_t n, size_t stride) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t window_stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n; i += window_stride) {
        size_t target_idx = (i + 1) * (2 * stride) - 1;

        if (target_idx < n) { 
            double tmp = data_d[target_idx - stride];
            data_d[target_idx - stride] = data_d[target_idx];
            data_d[target_idx] += tmp;
        }
    }
}


void seg_scan(double* data_d, size_t n, double* sum) {
    // pad the array
    size_t old_n = n;
    size_t bytes = n * sizeof(double);
    size_t old_bytes = bytes;
    double* arr_d;
    if ((n & (n - 1)) != 0) {
        n = 1 << ((size_t)log2(n) + 1);
        bytes = n * sizeof(bytes);
    }
    
    CHECK_CUDA(hipMalloc(&arr_d, bytes));
    CHECK_CUDA(hipMemcpy(arr_d, data_d, old_bytes, hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemset(&arr_d[old_n], 0, bytes - old_bytes));

    for (size_t stride = 1; stride < n; stride *= 2) {
        seg_scan_red_kernel<<<num_blocks, threads_per_block>>>(arr_d, n, stride);
        CHECK_CUDA(hipGetLastError());
        
        CHECK_CUDA(hipDeviceSynchronize());
    }

    CHECK_CUDA(hipMemcpy(sum, &arr_d[n - 1], sizeof(double), hipMemcpyDeviceToHost));
    double zero = 0;
    CHECK_CUDA(hipMemcpy(&arr_d[n - 1], &zero, sizeof(double), hipMemcpyHostToDevice));

    for (size_t stride = n / 2; stride > 0; stride /= 2) {
        seg_scan_ds_kernel<<<num_blocks, threads_per_block>>>(arr_d, n, stride);
        CHECK_CUDA(hipGetLastError());
        
        CHECK_CUDA(hipDeviceSynchronize());
    }

    CHECK_CUDA(hipMemcpy(data_d, arr_d, old_bytes, hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipFree(arr_d));
}



int main(const int argc, const char** argv) {
    size_t N = 10;
    if (argc > 1) {
        N = atoi(argv[1]);
    }

    size_t bytes = N * sizeof(double);
    double *arr_neighbors_d, *arr_stride_d, *arr_seg_scan_d;

    CHECK_CUDA(hipMalloc(&arr_neighbors_d, bytes));
    CHECK_CUDA(hipMalloc(&arr_stride_d, bytes));
    CHECK_CUDA(hipMalloc(&arr_seg_scan_d, bytes));
    
    int deviceId;
	CHECK_CUDA(hipGetDevice(&deviceId));
	hipDeviceProp_t prop;
	CHECK_CUDA(hipGetDeviceProperties(&prop, deviceId));
	int sm = prop.multiProcessorCount;
    num_blocks = 2 * sm;
    threads_per_block = 4 * MIN_CUDA_THREADS;
    
    init_arr<<<num_blocks, threads_per_block>>>(arr_neighbors_d, N, time(NULL));
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(arr_stride_d, arr_neighbors_d, bytes, hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(arr_seg_scan_d, arr_neighbors_d, bytes, hipMemcpyDeviceToDevice));

    StartTimer();
    
    double neighbor = reduction_neighbor(arr_neighbors_d, N);
    
    const double t_neighbor = GetTimer() / 1000.0;
    
    StartTimer();

    double stride = reduction_stride(arr_stride_d, N);

    const double t_stride = GetTimer() / 1000.0;

    StartTimer();

    double sum;
    seg_scan(arr_seg_scan_d, N, &sum);
    
    const double t_scan = GetTimer() / 1000.0;
    printf("%zu,%lf,%lf,%lf,%lf,%lf,%lf\n", N, 
        neighbor, t_neighbor, stride, t_stride, sum, t_scan);
 
    CHECK_CUDA(hipFree(arr_neighbors_d));
    CHECK_CUDA(hipFree(arr_stride_d));

    return 0;
}

void print_arr(double* data, size_t N) {
    for (int i = 0; i < N; i++) {
        printf("%lf, ", data[i]);
    }
    printf("\n");
}

